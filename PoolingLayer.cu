#include "PoolingLayer.h"

PoolingLayer::PoolingLayer(hipdnnHandle_t handle): handle(handle) {
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pooling_descriptor));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
}

void PoolingLayer::SetInputDescriptor(hipdnnTensorDescriptor_t prev_output_descriptor) {
    input_descriptor = prev_output_descriptor;
}

void PoolingLayer::SetInputData(float* data) {
    input_data = data;
}

void PoolingLayer::SetPoolingDescriptor(int window_H, int window_W, int stride_V, int stride_H) {
    window_height = window_H;
    window_width = window_W;
    stride_vertical = stride_V;
    stride_horizontal = stride_H;

    CUDNN_CALL(hipdnnSetPooling2dDescriptor(pooling_descriptor,
                                           HIPDNN_POOLING_MAX,
                                           HIPDNN_NOT_PROPAGATE_NAN,
                                           window_height,
                                           window_width,
                                           /*Pad H*/0,
                                           /*Pad W*/0,
                                           stride_vertical,
                                           stride_horizontal));
}

void PoolingLayer::SetOutputDescriptor(int N, int C, int H, int W) {
    CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(pooling_descriptor, 
                                                 input_descriptor,
                                                 &output_n, &output_c, &output_h, &output_w));

    printf("Pooling 1_Output Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", output_n, output_c, output_h, output_w);

    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor, 
                                          HIPDNN_TENSOR_NCHW, 
                                          HIPDNN_DATA_FLOAT,
                                          output_n, output_c, output_h, output_w));
}

float* PoolingLayer::GetOutputData() {
    return output_data;
}
 
void PoolingLayer::AllocateMemory() {
    CUDA_CALL(hipMalloc(&output_data, output_n * output_c * output_h * output_w * sizeof(float)));
}

void PoolingLayer::Forward() {
    CUDNN_CALL(hipdnnPoolingForward(handle,
                                   pooling_descriptor,
                                   &alpha,
                                   input_descriptor, input_data,
                                   &beta,
                                   output_descriptor, output_data));

    //hipFree(input_data);
}