#include "PoolingLayer.h"

PoolingLayer::PoolingLayer(hipdnnHandle_t handle): handle(handle) {
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor))
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pooling_descriptor));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
}

void PoolingLayer::SetInputDescriptor(int N, int C, int H, int W) {
    input_n = N;
    input_c = C;
    input_h = H;
    input_w = W;

    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor, 
                                          HIPDNN_TENSOR_NCHW, 
                                          HIPDNN_DATA_FLOAT,
                                          input_n, input_c, input_h, input_w));
    #if DEBUG
    printf("Pooling Input Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", input_n, input_c, input_h, input_w);
    #endif
}

void PoolingLayer::SetInputData(float* data) {
    input_data = data;
}

void PoolingLayer::SetPoolingDescriptor(int window_H, int window_W, int stride_V, int stride_H) {
    window_height = window_H;
    window_width = window_W;
    stride_vertical = stride_V;
    stride_horizontal = stride_H;

    CUDNN_CALL(hipdnnSetPooling2dDescriptor(pooling_descriptor,
                                           HIPDNN_POOLING_MAX,
                                           HIPDNN_NOT_PROPAGATE_NAN,
                                           window_height,
                                           window_width,
                                           /*Pad H*/0,
                                           /*Pad W*/0,
                                           stride_vertical,
                                           stride_horizontal));
}

void PoolingLayer::SetOutputDescriptor(int N, int C, int H, int W) {
    CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(pooling_descriptor, 
                                                 input_descriptor,
                                                 &output_n, &output_c, &output_h, &output_w));

    #if DEBUG
    printf("Pooling Output Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", output_n, output_c, output_h, output_w);
    #endif

    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor, 
                                          HIPDNN_TENSOR_NCHW, 
                                          HIPDNN_DATA_FLOAT,
                                          output_n, output_c, output_h, output_w));
}

float* PoolingLayer::GetOutputData() {
    return output_data;
}
 
void PoolingLayer::AllocateMemory() {
    CUDA_CALL(hipMalloc(&output_data, output_n * output_c * output_h * output_w * sizeof(float)));
}

void PoolingLayer::Forward() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    CUDNN_CALL(hipdnnPoolingForward(handle,
                                   pooling_descriptor,
                                   &alpha,
                                   input_descriptor, input_data,
                                   &beta,
                                   output_descriptor, output_data));
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("%f,", milliseconds);
}

void PoolingLayer::Free() {
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pooling_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDA_CALL(hipFree(input_data));
}