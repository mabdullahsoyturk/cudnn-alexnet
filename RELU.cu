#include "RELU.h"

RELU::RELU(hipdnnHandle_t handle, float *data) : 
        handle(handle), input_data(data) {
    
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnCreateActivationDescriptor(&activation_descriptor));
    CUDNN_CALL(hipdnnSetActivationDescriptor(activation_descriptor,
                                            HIPDNN_ACTIVATION_RELU,
                                            HIPDNN_PROPAGATE_NAN,
                                            /*RELU_coef=*/0));
}

void RELU::SetInputDescriptor(int N, int C, int H, int W) {
    input_n = N;
    input_c = C;
    input_h = H;
    input_w = W;

    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor, 
                                        HIPDNN_TENSOR_NCHW, 
                                        HIPDNN_DATA_FLOAT,
                                        input_n, input_c, input_h, input_w));
    
    #if DEBUG
    printf("RELU Input Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", input_n, input_c, input_h, input_w);
    #endif

    RELU::SetOutputDescriptor();

    #if DEBUG
    printf("RELU Output Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", output_n, output_c, output_h, output_w);
    #endif
}

void RELU::SetOutputDescriptor() {
    output_n = input_n;
    output_c = input_c;
    output_w = input_w;
    output_h = input_h;
}

void RELU::Forward() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    CUDNN_CALL(hipdnnActivationForward(handle,
                                      activation_descriptor,
                                      &alpha,
                                      input_descriptor,
                                      input_data,
                                      &beta,
                                      input_descriptor,
                                      input_data));
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("%f,", milliseconds);
    hipDeviceSynchronize();
}

void RELU::Free() {
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyActivationDescriptor(activation_descriptor));
}
