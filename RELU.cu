#include "RELU.h"

RELU::RELU(hipdnnHandle_t handle, hipdnnTensorDescriptor_t descriptor, float *input_data) : 
        handle(handle), input_descriptor(descriptor), data(input_data) {
    
    CUDNN_CALL(hipdnnCreateActivationDescriptor(&activation_descriptor));
    CUDNN_CALL(hipdnnSetActivationDescriptor(activation_descriptor,
                                            HIPDNN_ACTIVATION_RELU,
                                            HIPDNN_PROPAGATE_NAN,
                                            /*RELU_coef=*/0));
}

void RELU::Forward() {
    CUDNN_CALL(hipdnnActivationForward(handle,
                                      activation_descriptor,
                                      &alpha,
                                      input_descriptor,
                                      data,
                                      &beta,
                                      input_descriptor,
                                      data));
}