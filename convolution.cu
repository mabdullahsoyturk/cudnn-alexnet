#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils.h"
#include <hipDNN.h>

int main() {
  hipdnnHandle_t cudnn;
  CUDNN_CALL(hipdnnCreate(&cudnn));

  // Input
  const int input_n = 1;
  const int input_c = 3;
  const int input_h = 224;
  const int input_w = 224;
  printf("Input Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", input_n, input_c, input_h, input_w);

  hipdnnTensorDescriptor_t input_descriptor;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor, 
                                        HIPDNN_TENSOR_NCHW, 
                                        HIPDNN_DATA_FLOAT,
                                        input_n, input_c, input_h, input_w));

  float *input_data;
  CUDA_CALL(hipMalloc(&input_data, input_n * input_c * input_h * input_w * sizeof(float)));
  fill_constant<<<input_w * input_h, input_n * input_c>>>(input_data, 1.f);
  hipDeviceSynchronize();

  // Filter
  const int filter_n = 96;
  const int filter_c = 3;
  const int filter_h = 11;
  const int filter_w = 11;
  printf("Filter Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", filter_n, filter_c, filter_h, filter_w);

  hipdnnFilterDescriptor_t filter_descriptor;
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
  CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_descriptor, 
                                        HIPDNN_DATA_FLOAT, 
                                        HIPDNN_TENSOR_NCHW,
                                        filter_n, filter_c, filter_h, filter_w));

  float *filter_data;
  CUDA_CALL(hipMalloc(&filter_data, filter_n * filter_c * filter_h * filter_w * sizeof(float)));
  fill_constant<<<filter_w * filter_h, filter_n * filter_c>>>(filter_data, 1.f);
  hipDeviceSynchronize();

  // Convolution
  const int padding_h = 2;
  const int padding_w = 2;
  const int stride_h = 4;
  const int stride_w = 4;
  const int dilation_h = 1;
  const int dilation_w = 1;
  printf("Convolution parameters => Padding h: %d, Padding w: %d, Stride h: %d, Stride w: %d, Dilation h: %d, Dilation w: %d\n",
                                    padding_h,     padding_w,     stride_h,     stride_w,     dilation_h,     dilation_w);

  hipdnnConvolutionDescriptor_t convolution_descriptor;
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  CUDNN_CALL(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                             padding_h, padding_w, stride_h, stride_w, dilation_h, dilation_w,
                                             HIPDNN_CONVOLUTION, 
                                             HIPDNN_DATA_FLOAT));

  // Output
  int output_n, output_c, output_h, output_w;
  CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor, 
                                                   input_descriptor, filter_descriptor,
                                                   &output_n, &output_c, &output_h, &output_w));

  printf("Convolution 1 Output Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", output_n, output_c, output_h, output_w);

  hipdnnTensorDescriptor_t output_descriptor;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor, 
                                        HIPDNN_TENSOR_NCHW, 
                                        HIPDNN_DATA_FLOAT,
                                        output_n, output_c, output_h, output_w));

  float *output_data;
  CUDA_CALL(hipMalloc(&output_data, output_n * output_c * output_h * output_w * sizeof(float)));

  // Get the best algorithmThe first layer of AlexNet was a convolutional layer that accepted a (224×224×3) image tensor as its input. It performed a convolution operation using 96 (11×11) kernels with a stride of four and a padding of two. This produced a (55×55×96) output tensor that was then passed through a ReLu activation function then on to the next layer. The layer contained 34,944 trainable parameters.

  hipdnnConvolutionFwdAlgoPerf_t convolution_algo_perf;
  int algo_count;

  cudnnGetConvolutionForwardAlgorithm_v7(cudnn,
                                         input_descriptor,
                                         filter_descriptor,
                                         convolution_descriptor,
                                         output_descriptor,
                                         /*requested algo count*/1,
                                         /*returned algo count*/&algo_count,
                                         &convolution_algo_perf);
  
  hipdnnConvolutionFwdAlgo_t algorithm = convolution_algo_perf.algo;

  // Calculate how much workspace we need (Note: Not every algorithm needs workspace. If returns 0, that's okay)
  size_t workspace_size;
  CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, 
                                                     input_descriptor, filter_descriptor, convolution_descriptor, output_descriptor, 
                                                     algorithm, 
                                                     &workspace_size));

  float *workspace_data;
  CUDA_CALL(hipMalloc(&workspace_data, workspace_size));
  printf("Workspace allocated: %ld bytes\n", workspace_size);
  
  // Perform Convolution 1: 96 filters of 11x11x3 + 4 stride, padding: 2, output shape: 54x54x96
  float alpha = 1.f;
  float beta = 0.f;
  CUDNN_CALL(hipdnnConvolutionForward(cudnn,
                                     &alpha, 
                                     input_descriptor, input_data, 
                                     filter_descriptor, filter_data,
                                     convolution_descriptor, algorithm, workspace_data, workspace_size,
                                     &beta, 
                                     output_descriptor, output_data));

  // Perform Pooling 1: 3x3 max pooling + 2 stride, output shape: 27x27x96
  int window_height = 3;
  int window_width = 3;
  int pooling_stride_vertical = 2;
  int pooling_stride_horizontal = 2;

  hipdnnPoolingDescriptor_t pooling_descriptor;
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pooling_descriptor));
  CUDNN_CALL(hipdnnSetPooling2dDescriptor(pooling_descriptor,
                              HIPDNN_POOLING_MAX,
                              HIPDNN_NOT_PROPAGATE_NAN,
                              window_height,
                              window_width,
                              /*Pad H*/0,
                              /*Pad W*/0,
                              pooling_stride_vertical,
                              pooling_stride_horizontal));
  
  int pooling_output_n, pooling_output_c, pooling_output_h, pooling_output_w;
  CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(pooling_descriptor, 
                                               output_descriptor,
                                               &pooling_output_n, &pooling_output_c, &pooling_output_h, &pooling_output_w));

  printf("Pooling 1_Output Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", pooling_output_n, pooling_output_c, pooling_output_h, pooling_output_w);

  hipdnnTensorDescriptor_t pooling_output_descriptor;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&pooling_output_descriptor));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(pooling_output_descriptor, 
                                        HIPDNN_TENSOR_NCHW, 
                                        HIPDNN_DATA_FLOAT,
                                        pooling_output_n, pooling_output_c, pooling_output_h, pooling_output_w));

  float *pooling_output_data;
  // Perform Convolution 2: 256 filters of 5x5x3 + 1 stride, padding: 2, output shape: 27x27x256

  // Perform Pooling 2: 3x3 max pooling + 2 stride, output shape: 13x13x256

  // Perform Convolution 3: 384 filters of 13x13x3 + 1 stride, padding: 1, output shape: 13x13x384
  // Perform Convolution 4: 384 filters of 13x13x3 + 1 stride, padding: 1, output shape: 13x13x384
  // Perform Convolution 5: 256 filters of 3x3x3 + 1 stride, padding: 1, output shape: 13x13x256
  // Perform Pooling 3: 3x3 max pooling + 2 stride, output shape: 6x6x256

  // Results
 /*  std::cout << "input_data:" << std::endl;
  print(input_data, input_n, input_c, input_h, input_w);
  
  std::cout << "filter_data:" << std::endl;
  print(filter_data, filter_n, filter_c, filter_h, filter_w);
  
  std::cout << "output_data:" << std::endl;
  print(output_data, output_n, output_c, output_h, output_w);
 */
  // Clean your mess
  CUDA_CALL(hipFree(workspace_data));
  CUDA_CALL(hipFree(output_data));
  CUDA_CALL(hipFree(pooling_output_data));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
  CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(convolution_descriptor));
  CUDA_CALL(hipFree(filter_data));
  CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
  CUDA_CALL(hipFree(input_data));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
  CUDNN_CALL(hipdnnDestroy(cudnn));
  return 0;
}
