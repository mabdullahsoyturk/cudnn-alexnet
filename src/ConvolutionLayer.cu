#include "ConvolutionLayer.h"

ConvolutionLayer::ConvolutionLayer(hipdnnHandle_t handle, float* data): handle(handle), input_data(data) {
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
}

void ConvolutionLayer::SetInputDescriptor(int N, int C, int H, int W) {
    input_n = N;
    input_c = C;
    input_h = H;
    input_w = W;

    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor, 
                                          HIPDNN_TENSOR_NCHW, 
                                          HIPDNN_DATA_FLOAT,
                                          input_n, input_c, input_h, input_w));
    
    #if DEBUG
    printf("Convolution Input Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", input_n, input_c, input_h, input_w);
    #endif
}

void ConvolutionLayer::SetFilterDescriptor(int N, int C, int H, int W) {
    filter_n = N;
    filter_c = C;
    filter_h = H;
    filter_w = W;

    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_descriptor, 
                                    HIPDNN_DATA_FLOAT, 
                                    HIPDNN_TENSOR_NCHW,
                                    filter_n, filter_c, filter_h, filter_w));

    #if DEBUG
    printf("Convolution Filter Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", filter_n, filter_c, filter_h, filter_w);
    #endif
}

void ConvolutionLayer::SetOutputDescriptor() {
    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor, 
                                                     input_descriptor, filter_descriptor,
                                                     &output_n, &output_c, &output_h, &output_w));

    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor, 
                                          HIPDNN_TENSOR_NCHW, 
                                          HIPDNN_DATA_FLOAT,
                                          output_n, output_c, output_h, output_w));
    
    #if DEBUG
    printf("Convolution Output Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", output_n, output_c, output_h, output_w);
    #endif
}

hipdnnTensorDescriptor_t ConvolutionLayer::GetOutputDescriptor() {
    return output_descriptor;
}

float* ConvolutionLayer::GetOutputData() {
    return output_data;
}

void ConvolutionLayer::SetConvolutionDescriptor(int H_padding, int W_padding, int H_stride, int W_stride, int H_dilation, int W_dilation) {
    padding_h = H_padding;
    padding_w = W_padding;
    stride_h = H_stride;
    stride_w = W_stride;
    dilation_h = H_dilation;
    dilation_w = W_dilation;

    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                             padding_h, padding_w, stride_h, stride_w, dilation_h, dilation_w,
                                             HIPDNN_CONVOLUTION, 
                                             HIPDNN_DATA_FLOAT));
    
    #if DEBUG
    printf("Convolution parameters => Padding h: %d, Padding w: %d, Stride h: %d, Stride w: %d, Dilation h: %d, Dilation w: %d\n",
                                    padding_h,     padding_w,     stride_h,     stride_w,     dilation_h,     dilation_w);
    #endif
}

void ConvolutionLayer::SetAlgorithm() {
    hipdnnConvolutionFwdAlgoPerf_t convolution_algo_perf;
    int algo_count;

    cudnnGetConvolutionForwardAlgorithm_v7(handle,
                                           input_descriptor,
                                           filter_descriptor,
                                           convolution_descriptor,
                                           output_descriptor,
                                           /*requested algo count*/1,
                                           /*returned algo count*/&algo_count,
                                           &convolution_algo_perf);
    
    algorithm = convolution_algo_perf.algo;
    //algorithm = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
}

void ConvolutionLayer::AllocateMemory() {
    CUDA_CALL(hipMalloc(&filter_data, filter_n * filter_c * filter_h * filter_w * sizeof(float)));
    CUDA_CALL(hipMalloc(&output_data, output_n * output_c * output_h * output_w * sizeof(float)));

    fill_constant<<<filter_w * filter_h, filter_n * filter_c>>>(filter_data, 1.f);
    hipDeviceSynchronize();
}

void ConvolutionLayer::AllocateWorkspace() {
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(handle, 
                                                       input_descriptor, filter_descriptor, convolution_descriptor, output_descriptor, 
                                                       algorithm, 
                                                       &workspace_size));

    CUDA_CALL(hipMalloc(&workspace_data, workspace_size));
    #if DEBUG
    printf("Workspace allocated: %ld bytes\n", workspace_size);
    #endif
}

void ConvolutionLayer::Forward() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    CUDNN_CALL(hipdnnConvolutionForward(handle,
                                       &alpha, 
                                       input_descriptor, input_data, 
                                       filter_descriptor, filter_data,
                                       convolution_descriptor, algorithm, workspace_data, workspace_size,
                                       &beta, 
                                       output_descriptor, output_data));
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("%f,", milliseconds);
    hipDeviceSynchronize();
}

void ConvolutionLayer::Free() {
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(convolution_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDA_CALL(hipFree(input_data));
    CUDA_CALL(hipFree(filter_data));
    CUDA_CALL(hipFree(workspace_data));
}
