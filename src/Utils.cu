#include "hip/hip_runtime.h"
#include "Utils.h"

__global__ void fill_constant(float *px, float k) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    px[tid] = k; 
}

void print(const float *data, int n, int c, int h, int w) {
    std::vector<float> buffer(1 << 20);
    CUDA_CALL(hipMemcpy(buffer.data(), data, n * c * h * w * sizeof(float), hipMemcpyDeviceToHost));

    int a = 0;
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < c; ++j) {
            std::cout << "n=" << i << ", c=" << j << ":" << std::endl;
            for (int k = 0; k < h; ++k) {
                for (int l = 0; l < w; ++l) {
                    std::cout << std::setw(10) << std::right << buffer[a];
                    ++a;
                }
                std::cout << std::endl;
            }
        }
    }

    std::cout << std::endl;
}
