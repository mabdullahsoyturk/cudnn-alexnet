#include <stdio.h>
#include "utils.h"
#include <hipDNN.h>

int main() {
  // Init cuDNN
  hipdnnHandle_t cudnn;
  CUDNN_CALL(hipdnnCreate(&cudnn));

  // Input
  const int input_n = 1;
  const int input_c = 1;
  const int input_h = 5;
  const int input_w = 5;
  printf("Input Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", input_n, input_c, input_h, input_w);

  hipdnnTensorDescriptor_t input_descriptor;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(
                                        input_descriptor, 
                                        HIPDNN_TENSOR_NCHW, 
                                        HIPDNN_DATA_FLOAT,
                                        input_n, input_c, input_h, input_w));

  float *input_data;
  CUDA_CALL(hipMalloc(&input_data, input_n * input_c * input_h * input_w * sizeof(float)));

  // Filter
  const int filter_n = 1;
  const int filter_c = 1;
  const int filter_h = 2;
  const int filter_w = 2;
  printf("Filter Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", filter_n, filter_c, filter_h, filter_w);

  hipdnnFilterDescriptor_t filter_descriptor;
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
  CUDNN_CALL(hipdnnSetFilter4dDescriptor(
                                        filter_descriptor, 
                                        HIPDNN_DATA_FLOAT, 
                                        HIPDNN_TENSOR_NCHW,
                                        filter_n, filter_c, filter_h, filter_w));

  float *filter_data;
  CUDA_CALL(hipMalloc(&filter_data, filter_n * filter_c * filter_h * filter_w * sizeof(float)));

  // Convolution
  const int padding_h = 1;
  const int padding_w = 1;
  const int stride_h = 1;
  const int stride_w = 1;
  const int dilation_h = 1;
  const int dilation_w = 1;
  printf("Convolution parameters => Padding h: %d, Padding w: %d, Stride h: %d, Stride w: %d, Dilation h: %d, Dilation w: %d\n",
                                    padding_h,     padding_w,     stride_h,     stride_w,     dilation_h,     dilation_w);

  hipdnnConvolutionDescriptor_t convolution_descriptor;
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
                                             convolution_descriptor,
                                             padding_h, padding_w, stride_h, stride_w, dilation_h, dilation_w,
                                             HIPDNN_CONVOLUTION, 
                                             HIPDNN_DATA_FLOAT));

  // Output
  int output_n, output_c, output_h, output_w;
  CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
                                                   convolution_descriptor, 
                                                   input_descriptor, filter_descriptor,
                                                   &output_n, &output_c, &output_h, &output_w));

  printf("Output Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", output_n, output_c, output_h, output_w);

  hipdnnTensorDescriptor_t output_descriptor;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(
                                        output_descriptor, 
                                        HIPDNN_TENSOR_NCHW, 
                                        HIPDNN_DATA_FLOAT,
                                        output_n, output_c, output_h, output_w));

  float *output_data;
  CUDA_CALL(hipMalloc(&output_data, output_n * output_c * output_h * output_w * sizeof(float)));

  // Get the best algorithm
  hipdnnConvolutionFwdAlgoPerf_t convolution_algo_perf;
  int algo_count;

  cudnnGetConvolutionForwardAlgorithm_v7(
                                         cudnn,
                                         input_descriptor,
                                         filter_descriptor,
                                         convolution_descriptor,
                                         output_descriptor,
                                         /*requested algo count*/1,
                                         /*returned algo count*/&algo_count,
                                         &convolution_algo_perf);
  
  hipdnnConvolutionFwdAlgo_t algorithm = convolution_algo_perf.algo;

  // Calculate how much workspace we need (Note: Not every algorithm needs workspace. If returns 0, that's okay)
  size_t workspace_size;
  CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
                                                     cudnn, 
                                                     input_descriptor, filter_descriptor, convolution_descriptor, output_descriptor, 
                                                     algorithm, 
                                                     &workspace_size));

  float *workspace_data;
  CUDA_CALL(hipMalloc(&workspace_data, workspace_size));

  printf("Workspace allocated: %ld bytes\n", workspace_size);

  // Fill input and filter data
  fill_thread_id<<<input_w * input_h, input_n * input_c>>>(input_data);
  fill_constant<<<filter_w * filter_h, filter_n * filter_c>>>(filter_data, 1.f);
  hipDeviceSynchronize();
  
  // Perform convolution
  float alpha = 1.f;
  float beta = 0.f;
  CUDNN_CALL(hipdnnConvolutionForward(
                                     cudnn,
                                     &alpha, 
                                     input_descriptor, input_data, 
                                     filter_descriptor, filter_data,
                                     convolution_descriptor, algorithm, workspace_data, workspace_size,
                                     &beta, 
                                     output_descriptor, output_data));

  // Results
  std::cout << "input_data:" << std::endl;
  print(input_data, input_n, input_c, input_h, input_w);
  
  std::cout << "filter_data:" << std::endl;
  print(filter_data, filter_n, filter_c, filter_h, filter_w);
  
  std::cout << "output_data:" << std::endl;
  print(output_data, output_n, output_c, output_h, output_w);

  // Clean your mess
  CUDA_CALL(hipFree(workspace_data));
  CUDA_CALL(hipFree(output_data));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
  CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(convolution_descriptor));
  CUDA_CALL(hipFree(filter_data));
  CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
  CUDA_CALL(hipFree(input_data));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
  CUDNN_CALL(hipdnnDestroy(cudnn));
  return 0;
}
